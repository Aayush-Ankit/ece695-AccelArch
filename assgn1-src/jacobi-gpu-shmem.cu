#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "mvm_check.h"
#include "parser.h"

const int TILE_DIM = 64; //blockDim.x = TILE_DIM must be

  __global__
void jacobi_sigma (float *A, float *b, float *x, float *sigma, int input_size)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    printf("sigma kernel starts...\n");
    //printf ("x_prev=%f\n", x[i]);
    sigma[i] = 0;
    for (int j=0; j<(input_size/TILE_DIM)+1; j++) //divides sigma[i] compute into (input_size/TILE_DIM) chunks for shmem
    {
      // read data from Gmem - a thread reads all values in a column
      // threads concurrently access the same row - coalesced access to Gmem
      for (int k=0; k<TILE_DIM; k++)
        tile[i][k] = A[k*input_size + ((j*TILE_DIM) + i)]; // tile has A's part in tranposed form (to avoid shmem bank conflicts)

      __syncthreads(); //a thread within a block brings data for different threads

      for (int k=0; k<TILE_DIM; k++)
        if (i != k)
          sigma[i] += tile[k][i]*x[k];

      __syncthreads(); //a thread can't fetch new data unless all previous data in shmem for the block gets used
    }
    printf ("sigma_value=%f\n", sigma[i]);
    printf("sigma kernel ends...\n");
  }
}

__global__
void jacobi_xnext (float *A, float *b, float*x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    x[i] = (b[i]-sigma[i])/A[i*input_size+ i];
    //printf ("x_next=%f\n", x[i]);
  }
}

int main (int argc, char **argv){
    // check if any file specified
    if (argc < 2)
    {
      cout << "Usage: " << "Input file name required" << endl;
      return 1;
    }

    char filename[20] = "inputs/";
    int N, M;
    float *A, *b, *x;

    strcat(filename, argv[1]);
    parser (filename, &N, &M, &A, &b);

    // allocate and initalize x with zeros
    int input_size = N;
    int num_iter = M;
    x = (float*) malloc(input_size*sizeof(float));
    for (int i = 0; i<input_size; i++)
        x[i] = 0.0;

    // Print the values parsed
    cout << "input size: " << input_size << endl;
    cout << "num iter: " << num_iter << endl;
    /*cout << "A matrix: \n";
    for (int i=0; i<input_size; i++)
    {
        for (int j=0; j<input_size; j++)
            cout << A[i*input_size + j] << " ";
        cout << endl;
    }
    cout << "b bector: \n";
    for (int i=0; i<input_size; i++)
        cout << b[i] << " ";
    cout << endl;*/

    // allocate device memory
    float *d_A, *d_b, *d_x, *d_sigma;
    hipMalloc((void**) &d_A, input_size*input_size*sizeof(float)); //hipMalloc has to be 1d array
    hipMalloc((void**) &d_b, input_size*sizeof(float));
    hipMalloc((void**) &d_x, input_size*sizeof(float));
    hipMalloc((void**) &d_sigma, input_size*sizeof(float));

    // move data from host to device memory
    hipMemcpy(d_A, A, (input_size*input_size)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, input_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, input_size*sizeof(float), hipMemcpyHostToDevice);

    //launch cuda kernel
    int block_width;
    if(argc > 2)  block_width = atoi(argv[2]);
    else  block_width = 1<<6; // defalut is 64

    cout << "block size: " << block_width << endl;
    //num_iter = 2;
    for (int k=0; k<num_iter; k++)
    {
      cout << "Iteration: " << k << endl;
      jacobi_sigma <<<(input_size+block_width-1)/block_width,block_width>>>
        (d_A, d_b, d_x, d_sigma, input_size);

      jacobi_xnext <<<(input_size+block_width-1)/block_width,block_width>>>
        (d_A, d_b, d_x, d_sigma, input_size);

      hipDeviceSynchronize(); // stops host execution until kernel finishes
    }

    // move data from device to host memory
    hipMemcpy(x, d_x, input_size*sizeof(float), hipMemcpyDeviceToHost);

    // test results
    mvm_check (A, x, b, input_size);

    // deallocate device and host memory
    hipFree(d_A); hipFree(d_b); hipFree(d_x);
    free(A); free(b); free(x);

    return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "mvm_check.h"
#include "parser.h"

  __global__
void jacobi_sigma (float *A, float *b, float *x, float *sigma, int input_size)
{
  extern __shared__ float x_shared[];

  // TILE_DIM is shared mmeory size - equal to block_width
  int TILE_DIM = blockDim.x;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    //printf ("x_prev=%f\n", x[i]);
    sigma[i] = 0;
    for (int j=0; j<((input_size-1)/TILE_DIM)+1; j++) //divides sigma[i] compute into (input_size/TILE_DIM) chunks for shmem
    {
      x_shared[threadIdx.x] = x[j*TILE_DIM + threadIdx.x];

      __syncthreads();

      for (int k=0; k<TILE_DIM; k++)
      {
        if (i != j*TILE_DIM + k)
          sigma[i] += A[(j*TILE_DIM+k)*input_size + i]*x_shared[k];
      }

      __syncthreads();
    }
  //printf ("sigma_value=%f\n", sigma[i]);
  }
}

__global__
void jacobi_xnext (float *A, float *b, float*x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    x[i] = (b[i]-sigma[i])/A[i*input_size+ i];
    //printf ("x_next=%f\n", x[i]);
  }
}

int main (int argc, char **argv){
    // check if any file specified
    if (argc < 2)
    {
      cout << "Usage: " << "Input file name required" << endl;
      return 1;
    }

    char filename[20] = "inputs/";
    int N, M;
    float *A, *b, *x;

    strcat(filename, argv[1]);
    parser (filename, &N, &M, &A, &b);

    // allocate and initalize x with zeros
    int input_size = N;
    int num_iter = M;
    x = (float*) malloc(input_size*sizeof(float));
    for (int i = 0; i<input_size; i++)
        x[i] = 0.0;

    // Print the values parsed
    cout << "input size: " << input_size << endl;
    cout << "num iter: " << num_iter << endl;
    /*cout << "A matrix: \n";
    for (int i=0; i<input_size; i++)
    {
        for (int j=0; j<input_size; j++)
            cout << A[i*input_size + j] << " ";
        cout << endl;
    }
    cout << "b bector: \n";
    for (int i=0; i<input_size; i++)
        cout << b[i] << " ";
    cout << endl;*/

    // create transpose of A (for coalesced GPU access) - even if counted this
    // shouldn't be an overhead (once for many iterrations of kernel launches)
    float *At = (float*) malloc(input_size*input_size*sizeof(float));
    for (int i=0; i<input_size; i++)
      for (int j=0; j<input_size; j++)
        At[j*input_size+i] = A[i*input_size+j];

    // allocate device memory
    //hipProfilerStart();
    float *d_A, *d_b, *d_x, *d_sigma;
    hipMalloc((void**) &d_A, input_size*input_size*sizeof(float)); //hipMalloc has to be 1d array
    hipMalloc((void**) &d_b, input_size*sizeof(float));
    hipMalloc((void**) &d_x, input_size*sizeof(float));
    hipMalloc((void**) &d_sigma, input_size*sizeof(float));

    // move data from host to device memory
    hipMemcpy(d_A, At, (input_size*input_size)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, input_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, input_size*sizeof(float), hipMemcpyHostToDevice);

    //launch cuda kernel
    int block_width;
    if(argc > 2)  block_width = atoi(argv[2]);
    else  block_width = 1<<6; // defalut is 64

    cout << "block size: " << block_width << endl;
    //num_iter = 1;
    for (int k=0; k<num_iter; k++)
    {
      //cout << "Iteration " << k << endl;
      jacobi_sigma
        <<<(input_size+block_width-1)/block_width,block_width,block_width*sizeof(float)>>>
        (d_A, d_b, d_x, d_sigma, input_size);

      jacobi_xnext <<<(input_size+block_width-1)/block_width,block_width>>>
        (d_A, d_b, d_x, d_sigma, input_size);

      //hipDeviceSynchronize(); // stops host execution until kernel finishes
    }

    // move data from device to host memory
    hipMemcpy(x, d_x, input_size*sizeof(float), hipMemcpyDeviceToHost);
    //hipProfilerStop();

    // test results
    mvm_check (A, x, b, input_size);

    // deallocate device and host memory
    hipFree(d_A); hipFree(d_b); hipFree(d_x);
    free(A); free(b); free(x); free (At);

    return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "mvm_check.h"
#include "parser.h"

  __global__
void jacobi_sigma (float *A, float *b, float *x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    sigma[i] = 0;
    for (int j=0; j<input_size; j++)
    {
      if (i != j)
        sigma[i] += A[j*input_size+ i]*x[j];
    }
  printf ("x_prev=%f\n", x[i]);
  //printf ("sigma_value=%f\n", sigma[i]);
  }
}

__global__
void jacobi_xnext (float *A, float *b, float*x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    x[i] = (b[i]-sigma[i])/A[i*input_size+ i];
    //printf ("x_next=%f\n", x[i]);
  }
}

int main (int argc, char **argv){
    // check if any file specified
    if (argc < 2)
    {
      cout << "Usage: " << "Input file name required" << endl;
      return 1;
    }

    char filename[20] = "inputs/";
    int N, M;
    float *A, *b, *x;

    strcat(filename, argv[1]);
    parser (filename, &N, &M, &A, &b);

    // allocate and initalize x with zeros
    int input_size = N;
    int num_iter = M;
    x = (float*) malloc(input_size*sizeof(float));
    for (int i = 0; i<input_size; i++)
        x[i] = 0.0;

    // Print the values parsed
    cout << "input size: " << input_size << endl;
    cout << "num iter: " << num_iter << endl;
    /*cout << "A matrix: \n";
    for (int i=0; i<input_size; i++)
    {
        for (int j=0; j<input_size; j++)
            cout << A[i*input_size + j] << " ";
        cout << endl;
    }
    cout << "b bector: \n";
    for (int i=0; i<input_size; i++)
        cout << b[i] << " ";
    cout << endl;*/

    // allocate unified memory
    float *A_u, *b_u, *x_u, *sigma_u;
    hipMallocManaged(&A_u, input_size*input_size*sizeof(float));
    hipMallocManaged(&b_u, input_size*sizeof(float));
    hipMallocManaged(&x_u, input_size*sizeof(float));
    hipMallocManaged(&sigma_u, input_size*sizeof(float));

    // initialize allocated memories based on parsed values
    for (int i=0; i<input_size; i++)
    {
      b_u[i] = b[i];
      x_u[i] = x[i];
    }
    for (int i=0; i<input_size; i++)
      for (int j=0; j<input_size; j++)
        A_u[j*input_size+i] = A[i*input_size+j];

    // deallocate memory associated with parsed variables
    free (b); free (x);

    //launch cuda kernel
    int block_width;
    if(argc > 2)  block_width = atoi(argv[2]);
    else  block_width = 1<<6; // defalut is 64

    cout << "block size: " << block_width << endl;
    //num_iter = 1;
    for (int k=0; k<num_iter; k++)
    {
      jacobi_sigma <<<(input_size+block_width-1)/block_width,block_width>>>
        (A_u, b_u, x_u, sigma_u, input_size);

      jacobi_xnext <<<(input_size+block_width-1)/block_width,block_width>>>
        (A_u, b_u, x_u, sigma_u, input_size);
    }

    // test results - in host
    hipDeviceSynchronize(); // stops host execution until kernel finishes
    mvm_check (A, x_u, b_u, input_size); //using non-transpose matrix for A for check

    // deallocate device and host memory
    hipFree(A_u); hipFree(b_u); hipFree(x_u); hipFree(sigma_u);
    free(A);

    //hipProfilerStop();
    return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "mvm_check.h"
#include "parser.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

  __global__
void jacobi_sigma (float *A, float *b, float *x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    //printf ("x_prev=%f\n", x[i]);
    sigma[i] = 0;
    for (int j=0; j<input_size; j++)
    {
      if (i != j)
      {
        sigma[i] += A[i*input_size+ j]*x[j];
        //if (j == 0) //see uncoalesced access
        //  printf ("A addr %u\n", &A[i*input_size+j]);
      }
    }
  //printf ("sigma_value=%f\n", sigma[i]);
  }
}

__global__
void jacobi_xnext (float *A, float *b, float*x, float *sigma, int input_size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < input_size)
  {
    x[i] = (b[i]-sigma[i])/A[i*input_size+ i];
    //printf ("x_next=%f\n", x[i]);
  }
}

int main (int argc, char **argv){
    // check if any file specified
    if (argc < 2)
    {
      cout << "Usage: " << "Input file name required" << endl;
      return 1;
    }

    char filename[20] = "inputs/";
    int N, M;
    float *A, *b, *x;

    strcat(filename, argv[1]);
    parser (filename, &N, &M, &A, &b);

    // allocate and initalize x with zeros
    int input_size = N;
    int num_iter = M;
    x = (float*) malloc(input_size*sizeof(float));
    for (int i = 0; i<input_size; i++)
        x[i] = 0.0;

    // Print the values parsed
    cout << "input size: " << input_size << endl;
    cout << "num iter: " << num_iter << endl;
    /*cout << "A matrix: \n";
    for (int i=0; i<input_size; i++)
    {
        for (int j=0; j<input_size; j++)
            cout << A[i*input_size + j] << " ";
        cout << endl;
    }
    cout << "b bector: \n";
    for (int i=0; i<input_size; i++)
        cout << b[i] << " ";
    cout << endl;*/

    // allocate unified memory
    float *A_u, *b_u, *x_u, *sigma_u;
    hipMallocManaged(&A_u, input_size*input_size*sizeof(float));
    hipMallocManaged(&b_u, input_size*sizeof(float));
    hipMallocManaged(&x_u, input_size*sizeof(float));
    hipMallocManaged(&sigma_u, input_size*sizeof(float));

    // initialize allocated memories based on parsed values
    for (int i=0; i<input_size; i++)
    {
      b_u[i] = b[i];
      x_u[i] = x[i];
    }
    for (int i=0; i<input_size*input_size; i++)
      A_u[i] = A[i];

    // deallocate memory associated with parsed variables
    free (A); free (b); free (x);

    //launch cuda kernel
    int block_width;
    if(argc > 2)  block_width = atoi(argv[2]);
    else  block_width = 1<<6; // defalut is 64

    cout << "block size: " << block_width << endl;
    //num_iter = 1;
    for (int k=0; k<num_iter; k++)
    {
      //cout << "Iteration " << k << endl;
      jacobi_sigma <<<(input_size+block_width-1)/block_width,block_width>>>
        (A_u, b_u, x_u, sigma_u, input_size);

      jacobi_xnext <<<(input_size+block_width-1)/block_width,block_width>>>
        (A_u, b_u, x_u, sigma_u, input_size);

      //hipDeviceSynchronize(); // stops host execution until kernel finishes
    }

    // test results - in host
    hipDeviceSynchronize(); // stops host execution until kernel finishes
    mvm_check (A_u, x_u, b_u, input_size);

    // deallocate device and host memory
    hipFree(A_u); hipFree(b_u); hipFree(x_u);
    //hipProfilerStop();

    return 0;
}
